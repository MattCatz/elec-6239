#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "term.h"

// This is the default block size
const unsigned int BLOCK_SIZE = 32;

__global__ void convolution_basic(matrix_t* image, matrix_t* filter, matrix_t* result) {
    matrix_t sum;
    int rowd,cold,Bx,By,Tx,Ty;
    int i,j,half,start_x,start_y,end_x,end_y;

    Bx = blockIdx.x;
    By = blockIdx.y;
    Tx = threadIdx.x;
    Ty = threadIdx.y;

    sum = 0;

    rowd = By * BLOCK_SIZE + Ty;
    cold = Bx * BLOCK_SIZE + Tx;

    half = (W_SMOOTHING - 1)/2;

    start_y = rowd - half >= 0 ? -half : -rowd;
    end_y = rowd + half < M ? half : M - rowd - 1;
    
    for(i=start_y; i <= end_y; ++i) {
        start_x = cold - half >= 0 ? -half :  - cold;
        end_x = cold + half < M ? half : M - cold - 1;
        for(j=start_x; j <= end_x; ++j) {
            sum += image[M*(rowd+i)+(cold+j)]*filter[W_SMOOTHING*(i+half)+(j+half)];
        }
    }

    result[M*(rowd) + cold] = sum;
}

__global__ void convolution_edge(matrix_t* image, matrix_t* filter, matrix_t* result) {
    matrix_t gx,gy;
    int rowd,cold,Bx,By,Tx,Ty;
    int i,j,half,start_x,start_y,end_x,end_y;

    Bx = blockIdx.x;
    By = blockIdx.y;
    Tx = threadIdx.x;
    Ty = threadIdx.y;

    gx = 0;
    gy = 0;

    rowd = By * BLOCK_SIZE + Ty;
    cold = Bx * BLOCK_SIZE + Tx;

    half = (W_EDGE - 1)/2;

    start_y = rowd - half >= 0 ? -half : -rowd;
    end_y = rowd + half < M ? half : M - rowd - 1;

    for(i=start_y; i <= end_y; ++i) {
        start_x = cold - half >= 0 ? -half :  - cold;
        end_x = cold + half < M ? half : M - cold - 1;
        for(j=start_x; j <= end_x; ++j) {
            gx += image[M*(rowd+i)+(cold+j)]*filter[W_EDGE*(i+half)+(j+half)];
            gy += image[M*(rowd+i)+(cold+j)]*filter[W_EDGE*(j+half)+(i+half)];
        }
    }

    result[M*(rowd) + cold] = sqrt(gx*gx+gy*gy) > 70 ? 255 : 0;
}

int main(void) {

    // Pointers for host
    matrix_t *image;
    matrix_t *gaussian;
    matrix_t *sobel;
    matrix_t *result;

    // Pointers for device memory
    matrix_t *image_d;
    matrix_t *gaussian_d;
    matrix_t *sobel_d;
    matrix_t *result_d;

    // Used to measure performance
    hipEvent_t start, stop;

    // Used for timing
    float msecTotal = 0.0f;

    printf("Using block size %d\n", BLOCK_SIZE);

    // Allocate host memory for matrices
    // We have to cast our calloc/malloc
    // because cuda is technically a subset
    // of c++ not vanilla c 
    image = (matrix_t *) calloc(M*M, sizeof(matrix_t));
    assert(image != NULL);

    gaussian = (matrix_t *) calloc(W_SMOOTHING*W_SMOOTHING, sizeof(matrix_t));
    assert(gaussian != NULL);

    sobel = (matrix_t *) calloc(W_EDGE*W_EDGE, sizeof(matrix_t));
    assert(sobel != NULL);

    result = (matrix_t *) calloc(M*M, sizeof(matrix_t));
    assert(result != NULL);

    get_image(image);
    generate_guassian_2d(gaussian);
    generate_sobel_2d(sobel);

    save_ppm("Leaves_original_cuda.ppm", image);
    save_g("2d", gaussian);

    // Allocate device memory for matricies
    checkCudaErrors(hipMalloc((void **) &(image_d), M*M*sizeof(matrix_t)));
    checkCudaErrors(hipMalloc((void **) &(gaussian_d), W_SMOOTHING*W_SMOOTHING*sizeof(matrix_t)));
    checkCudaErrors(hipMalloc((void **) &(sobel_d), W_EDGE*W_EDGE*sizeof(matrix_t)));
    checkCudaErrors(hipMalloc((void **) &(result_d), M*M*sizeof(matrix_t)));

    // Copy host memory to device
    checkCudaErrors(hipMemcpy(image_d, image, M*M*sizeof(matrix_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(gaussian_d, gaussian,  W_SMOOTHING*W_SMOOTHING*sizeof(matrix_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(sobel_d, sobel,  W_EDGE*W_EDGE*sizeof(matrix_t), hipMemcpyHostToDevice));

    // Allocate CUDA events that we'll use for timing
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // Setup execution parameters TODO
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(M / threads.x, M / threads.y);

    printf("Computing result...\n");

    hipDeviceSynchronize();

    // Record the start event
    checkCudaErrors(hipEventRecord(start, NULL));

    // Execute the kernel
    convolution_basic <<< grid, threads >>>(image_d, gaussian_d, result_d);
    checkCudaErrors(hipMemcpy(image_d, result_d, M*M*sizeof(matrix_t), hipMemcpyDefault));
    convolution_edge <<< grid, threads >>>(image_d, sobel_d, result_d);

    // Copy result from device to host
    checkCudaErrors(hipMemcpy(result, result_d, M*M*sizeof(matrix_t), hipMemcpyDeviceToHost));

    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, NULL));
    checkCudaErrors(hipEventSynchronize(stop));

    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    printf("done in % 3f (sec)\n", msecTotal/1000);

    save_ppm("Leaves_blur_cuda.ppm", result);

    // Clean up memory
    free(image);
    free(gaussian);
    free(sobel);
    checkCudaErrors(hipFree(image_d));
    checkCudaErrors(hipFree(gaussian_d));
    checkCudaErrors(hipFree(sobel_d));

    /* end multiplication */

    return 0;
}
